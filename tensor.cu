
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>

using namespace std;

class Net {
private:
	int rows, cols;
	int** inputTensor;
	int** outputTensor;
public:
	Net(int, int);
	int relulayer();
	void poolingFunction(int, int, bool);
	void filterConvolve(int);
	void printArray();

};

Net::Net(int r, int c) {
	rows = r;
	cols = c;
	cout << "\n\nInitialization of " << rows << " x " << cols
			<< " 2D array with random values between -9 and 9\n\n";
	//input tensor initialization
	inputTensor = new int*[rows];
	for (int k = 0; k < rows; k++)
		inputTensor[k] = new int[cols];

	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			inputTensor[i][j] = rand() % 19 + (-9);
		}
	}

	//output tensor initialization to zero
	outputTensor = new int*[rows];
	for (int k = 0; k < rows; k++)
		outputTensor[k] = new int[cols];

	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			outputTensor[i][j] = 0;
		}
	}

}

int Net::relulayer() {
	/*
	 * A `ReLULayer` is the identity function for
	 * non-negative inputs, and gives zero for negative inputs.
	 */
	cout
			<< "\n\nReluLayer Identity Function converts negative inputs to zero\n\n";
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			if (inputTensor[i][j] < 0)
				outputTensor[i][j] = 0;
			else
				outputTensor[i][j] = inputTensor[i][j];
		}
	}
	return 0;
}

void Net::printArray() {

	cout << "\n\nINPUT TENSOR\n\n";
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			cout << inputTensor[i][j] << '\t';
		}
		cout << "\n";
	}

	cout << "\n\nOUTPUT TENSOR\n\n";
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			cout << outputTensor[i][j] << '\t';
		}
		cout << "\n";
	}
}

void Net::poolingFunction(int stride, int windowSize, bool minMax)
{

	//output matrix calculation
	//bool minMax=true implied min else max operation
	cout << "\n\nPooling function -- find on min/max operations\n\n";
	if (minMax == true)
		cout << "Pools using min operation, stride is " << stride
				<< " and window size is " << windowSize << endl;
	else
		cout << "Pools using max operation, stride is " << stride
				<< " and window size is " << windowSize << endl;


	int kernelSize = windowSize / 2;
	int minimum=999,maximum=-999;
	for (int y = 1; y < rows - 1; y++) {
		for (int x = 1; x < cols - 1; x++) {
			for (int k = -kernelSize; k <= kernelSize; k++) {
				for (int j = -kernelSize; j <= kernelSize; j++) {
					if (minMax == true)
						minimum = min(inputTensor[j + 1][k + 1],
								inputTensor[y - j][x - k]);
					else
						maximum = max(inputTensor[j + 1][k + 1],
								inputTensor[y - j][x - k]);
				}
			}
			if (minMax == true)
				outputTensor[y][x] = minimum;
			else
				outputTensor[y][x] = maximum;

		}
	}
}

void Net::filterConvolve(int windowSize) {
	cout << "\n\nConvolution function \n\n";
	int sum;
	int kernelSize = windowSize / 2;
	for (int y = 1; y < rows - 1; y++) {
		for (int x = 1; x < cols - 1; x++) {
			sum = 0;
			for (int k = -kernelSize; k <= kernelSize; k++) {
				for (int j = -kernelSize; j <= kernelSize; j++) {
					sum = sum
							+ inputTensor[j + 1][k + 1]
									* inputTensor[y - j][x - k];
				}
			}
			outputTensor[y][x] = sum;
		}
	}
}

int main() {
	Net net(3, 6);
	net.printArray();
	net.relulayer();
	net.printArray();
	net.poolingFunction(3, 3, false);
	net.printArray();
	net.filterConvolve(3);
	net.printArray();

	return 0;
}

